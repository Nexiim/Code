#include "hip/hip_runtime.h"
//
// Created by kevin on 21/03/23.
//

#include "grille.cuh"



/*Fonction lié à la grille*/
Grille::Grille(int witdh, int height, typeCellule c) : Graphe(witdh * height, c) {
    this->height = height;
    this->width = witdh;
    this->c =c;
    this->b = Bordure::BORD;
}

Grille::Grille(int witdh, int height, typeCellule c, double lambda) : Graphe(witdh * height, c, lambda) {
    this->height = height;
    this->width = witdh;
    this->c =c;
    this->b = Bordure::BORD;
}

Grille::Grille(int witdh, int height, typeCellule c, double lambda, double* precalcul) : Graphe(witdh * height, c, lambda, precalcul) {
    this->height = height;
    this->width = witdh;
    this->c =c;
    this->b = Bordure::BORD;
}

Cellule* Grille::getCellule(int i, int j) {
    return Graphe::getCellule((j * this->width) + i);
}



void Grille::reset(){
    for (int i = 0; i < this->width; i++) {
        for (int j = 0; j < this->height; j++) {    
                this->getCellule(i, j)->setEtat(Etat::NORMAL);
        }
    }
    
}

void Grille::affichageGrille() {
    for (int j = 0; j < this->height; j++) {
        for (int i = 0; i < this->width; i++) {
            std::cout << " " << this->getCellule(i, j)->getEtat();
        }
        std::cout << "\n";
    }
    std::cout << "\n";
}

int Grille::getWidth() {
    return this->width;
}

int Grille::getHeight() {
    return this->height;
}

void Grille::setBordure(Bordure b) {
    this->b = b;
}

typeVoisinage Grille::getVosinage(){
    return this->t;
}


void Grille::setVoisinage(typeVoisinage t){
    this->t = t;
    if (t == typeVoisinage::MOORE8) this->Moore8();
    else if (t == typeVoisinage::MOORE9) this->Moore9();
    else if (t == typeVoisinage::TOOM) this->Toom();
    else if (t == typeVoisinage::VONNEUMAN) this->VonNeumann();
    else if (t == typeVoisinage::VONNEUMAN4) this->VonNeumann4();
    else if (t == typeVoisinage::IRREGULIER) this->setVoisinageNorme(1);
    else {
        this->t = static_cast<typeVoisinage>(NULL);
        cout << " erreur setup voisinage";
    }
    setVoisinCUDA();
}

void Grille::setVoisinage(typeVoisinage t,double paramVoisinage){
    this->t = t;
    if (t == typeVoisinage::IRREGULIER) this->setVoisinageNorme(paramVoisinage);
    else {
        this->t = static_cast<typeVoisinage>(NULL);
        cout << " erreur setup voisinage";
    }
}


void Grille::setVoisinageNorme(double r) {
    nbVoisinMax = r*r;
    int Sizetab =25;
    int tab[Sizetab] ={0};
    cout << "voisinage circulaire" << endl;
    this->t = typeVoisinage::IRREGULIER;

    pair<double,double> points[this->getWidth()][this->getHeight()];

    for (int i = 0; i < this->getWidth(); i++){
        for (int j = 0; j <this->getHeight(); j++){
            points[i][j] = make_pair((((double) rand() / (RAND_MAX))-0.5),((double)rand() / (RAND_MAX))-0.5);
        }
    }
    double nbVoisinMoy =0;
    for (int i = 0; i < this->getWidth(); i++){
       for (int j = 0; j <this->getHeight(); j++){

            for (int  a = floor(i-r); a <= ceil(i+r); a++){
                for (int b = floor(j-r); b <= ceil(j+r); b++){
                    //cout << "a :" << a << "b :" << b << endl;
                    if (this->b == Bordure::BORD){
                        if (a >= 0 && a < this->getWidth() && b >=0 && b < this->getHeight()) {
                            /*on vérifie qu'on est dans le carrée inscrit dans le cercle
                            on utilise 1.5 mais il faut simplement une valeur supérieur sqrt(2)*/
                            if (abs(i - a) < floor(r)/1.5-1 & abs(j - b) < floor(r)/1.5-1) {
                                this->getCellule(i, j)->addVoisin(this->getCellule(a, b));
                            } else {
                            double distX = (i + points[i][j].first) - (a + points[a][b].first);
                            double distY = (j + points[i][j].second) - (b + points[a][b].second);
                            //cout << "first "<< i+points[i][j].first  << " " << a+points[a][b].first;
                            //cout << "second "<< j+points[i][j].second  << " " << b+points[a][b].second;

                            //cout << distX << " " << distY << endl;
                            double distEcuclidienne = sqrt(pow(distX, 2) + pow(distY, 2));
                            //cout << distEcuclidienne << endl;
                                if (distEcuclidienne < r) {
                                this->getCellule(i, j)->addVoisin(this->getCellule(a, b));
                                    //cout << "addVoisin" << endl;
                                }
                            }

                        }
                    }
                    else if(this->b == Bordure::TORIQUE){
                        int amod = (a+this->getWidth()) % this->getWidth() ;
                        int bmod = (b+this->getHeight()) % this->getHeight();
                       
                         double distX = abs((i + points[i][j].first) - (amod + points[amod][bmod].first));
                        distX = min(distX,this->getWidth()-distX);

                        double distY = abs((j + points[i][j].second) - (bmod + points[amod][bmod].second));
                        distY = min(distY,this->getHeight()-distY);

                        double distEcuclidienne = sqrt(pow(distX, 2) + pow(distY, 2));
                        //cout << distEcuclidienne << endl;
                        if (distEcuclidienne < r) {
                            this->getCellule(i, j)->addVoisin(this->getCellule(amod, bmod));
                        }
                    }
                    else{
                        //assert("setVoisinageNorme:bordure non explicite");
                    }

                }
            }
            int nbVoisin = this->getCellule(i, j)->nbVoisin();
            tab[nbVoisin] = tab[nbVoisin]+1;
            nbVoisinMoy = nbVoisinMoy + nbVoisin;
        }
    }

    nbVoisinMoy = nbVoisinMoy/(this->getHeight()*this->getWidth());
    cout << "nombre de voisin en moyenne :" << nbVoisinMoy<< endl;

    for (int i = 0; i < Sizetab; i++){
        cout << "tab["<< i << "]: " << tab[i] << endl;
    }

}

void Grille::setVoisinageNorme(int x, int y ,double r){
    nbVoisinMax = r*r;
    this->getCellule(x, y)->resetVoisin();
    pair<double,double> points[(int)(2*ceil(r))][(int)(2*ceil(r))];
    for (int i = 0; i < 2*ceil(r); i++){
        for (int j = 0; j < 2*ceil(r); j++){
            points[i][j] = make_pair((((double) rand() / (RAND_MAX))-0.5),((double)rand() / (RAND_MAX))-0.5);
        }
    }


    for (int i = -ceil(r); i < ceil(r); i++){
        for (int j = -ceil(r); j < ceil(r); j++){
            int cX = ceil(r);
            int cY = ceil(r);

            if ( cX+i >= 0 && cX+i < this->getWidth() && cY+j >=0 && cY+j < this->getHeight()) {

                double distX = x + points[cX][cY].first - x+i + points[cX+i][cY+j].first;
                double distY = y + points[cX][cY].second - y+j+ points[cX+i][cY+j].second;
                double distEcuclidienne = sqrt(pow(distX, 2) + pow(distY, 2));
                if (distEcuclidienne < r) {
                    this->getCellule(x, y)->addVoisin(this->getCellule(x+i, y+j));
                }
            }
        }
    }

}

void Grille::Moore8() {
    nbVoisinMax = 8;
    for (int i = 0; i < this->width; i++) {
        for (int j = 0; j < this->height; j++) {
            this->getCellule(i,j)->setNbVoisinMax(typeVoisinage::MOORE8);
            for (int a = -1; a < 2; a++) {
                for (int b = -1; b < 2; b++) {
                    if (a != 0 || b != 0) {
                        if(this->b == Bordure::TORIQUE) {
                            this->getCellule(i, j)->addVoisin(this->getCellule((this->width+(i + a))% this->width,
                                                                               (this->height+(j + b))% this->height));
                        }
                        else if(this->b == Bordure::BORD) {
                            if (((i + a) >= 0) & ((i + a) <= (this->width - 1)) & ((j + b) >= 0) &
                                ((j + b) <= (this->height - 1))) {
                                this->getCellule(i, j)->addVoisin(this->getCellule(i + a, j + b));
                            }
                        }
                    }
                }
            }
        }
    }
}

void Grille::Moore9() {
    nbVoisinMax = 9;
    for (int i = 0; i < this->width; i++) {
        for (int j = 0; j < this->height; j++) {
            this->getCellule(i,j)->setNbVoisinMax(typeVoisinage::MOORE9);
            for (int a = -1; a < 2; a++) {
                for (int b = -1; b < 2; b++) {
                    if (((i + a) >= 0) & ((i + a) <= (this->width - 1)) & ((j + b) >= 0) &
                        ((j + b) <= (this->height - 1))) {
                        this->getCellule(i, j)->addVoisin(this->getCellule(i + a, j + b));
                    }
                }
            }
        }
    }
}

void Grille::VonNeumann() {
    nbVoisinMax = 5;
    for (int i = 0; i < this->width; i++) {
        for (int j = 0; j < this->height; j++) {
            this->getCellule(i,j)->setNbVoisinMax(typeVoisinage::VONNEUMAN);
            this->getCellule(i, j)->addVoisin(this->getCellule(i, j));
            for (int a = -1; a < 2; a++) {
                for (int b = -1; b < 2; b++) {
                    if (a != b & a != -b) {
                        if(this->b == Bordure::TORIQUE){
                            this->getCellule(i, j)->addVoisin(this->getCellule((this->width+(i + a))% this->width,
                                                                               (this->height+(j + b))% this->height));
                        }
                        else if(this->b == Bordure::BORD) {
                            if (((i + a) >= 0) & ((i + a) <= (this->width - 1)) & ((j + b) >= 0) &
                                ((j + b) <= (this->height - 1))) {
                                this->getCellule(i, j)->addVoisin(this->getCellule(i + a, j + b));
                            }
                        }
                    }
                }
            }
        }
    }
}

void Grille::VonNeumann4() {
    nbVoisinMax = 4;
    for (int i = 0; i < this->width; i++) {
        for (int j = 0; j < this->height; j++) {
            this->getCellule(i,j)->setNbVoisinMax(typeVoisinage::VONNEUMAN4);
            for (int a = -1; a < 2; a++) {
                for (int b = -1; b < 2; b++) {
                    if (a != b & a != -b) {
                        if(this->b == Bordure::TORIQUE){
                            this->getCellule(i, j)->addVoisin(this->getCellule((this->width+(i + a))% this->width,
                                                                               (this->height+(j + b))% this->height));
                        }
                        else if(this->b == Bordure::BORD) {
                            if (((i + a) >= 0) & ((i + a) <= (this->width - 1)) & ((j + b) >= 0) &
                                ((j + b) <= (this->height - 1))) {
                                this->getCellule(i, j)->addVoisin(this->getCellule(i + a, j + b));
                            }
                        }
                    }
                }
            }
        }
    }
}

void Grille::Toom() {
    nbVoisinMax = 3;
    for (int i = 0; i < this->width; i++) {
        for (int j = 0; j < this->height; j++) {
            if(this->b == Bordure::TORIQUE) {
                this->getCellule(i, j)->addVoisin(this->getCellule((i + 1)%this->width, j));
                this->getCellule(i, j)->addVoisin(this->getCellule(i, (j+1)% this->height));
            }
            else if(this->b == Bordure::BORD) {
                this->getCellule(i, j)->setNbVoisinMax(typeVoisinage::TOOM);
                this->getCellule(i, j)->addVoisin(this->getCellule(i, j));
                if (i + 1 < this->width) this->getCellule(i, j)->addVoisin(this->getCellule(i + 1, j));
                if (j - 1 >= 0) this->getCellule(i, j)->addVoisin(this->getCellule(i, j - 1));
            }
        }
    }
}
