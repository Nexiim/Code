#include "hip/hip_runtime.h"
//
// Created by kevin on 14/03/23.
//

#include "graphe.cuh"

/*
Constructor
*/

Graphe::Graphe(int size, typeCellule c) {
    this->c = c;
    this->width = size*5;
    this->height = size*5;

    this->size = size;
    def = 0;

    cels = (Cellule**)malloc(size*sizeof(Cellule*));
    setupCUDA();

    for (int i = 0; i < size; i++) {
        if (c == typeCellule::DEFAULT) cels[i] = new Cellule();
        if (c == typeCellule::QUOROMD) cels[i] = new CelluleQuoromD();
        //if (c == typeCellule::CORRECTEUR) this->listeCellules.push_back(new Correcteur());    

        posCellule.push_back(make_pair(rand()%this->width,rand()%this->height));    
    }
    this->threshold = 0;
}

Graphe::Graphe(int size, typeCellule c, double lambda) {
    this->c = c;
    for (int i = 0; i < size; i++) {
        if (c == typeCellule::QUOROMD) cels[i] = new CelluleQuoromD(lambda);
    }
    this->threshold = 0;
}

Graphe::Graphe(int size, typeCellule c, double lambda, double* precalcul) {
    this->c = c;
    for (int i = 0; i < size; i++) {
        if (c == typeCellule::QUOROMD) cels[i] = new CelluleQuoromD(lambda, precalcul);
    }
    this->threshold = 0;
}



/*
fonction
*/

/*void Graphe::reset(){

}*/

void Graphe::reset(double proba, double lambda, double *precalcul) {
    posCellule.clear();

    for (int i = 0; i < nbCellule(); i++) {
        posCellule.emplace_back(rand()%this->width,rand()%this->height);
    }
    def = 0;
    for (int i = 0; i < nbCellule(); i++) {
            auto* cell = dynamic_cast<CelluleQuoromD*>(getCellule(i));
            cell->setLambda(lambda);
            cell->setPreCalcul(precalcul);
            double r = rand() / double(RAND_MAX);
            if (r < proba) {
                getCellule(i)->setEtat(DEFAILANTE);
                def++;
            }
            else getCellule(i)->setEtat(NORMAL);
    }
    threshold = 0;

    if (getVoisinage() == typeVoisinage::PLUSPROCHE)
        resetVoisinPlusProche();
}


void Graphe::setupCUDA(){
    hipMalloc(&d_cels, size*sizeof(int));
    hipMalloc(&d_celsSync, size*sizeof(int));
    hipMalloc(&d_nbAlerte, sizeof(int));

    alerte = (int*)malloc(sizeof(int));
    *alerte = 0;
}

void Graphe::setVoisinCUDA(){
    int *pos = (int*)malloc(nbCellule()*2*sizeof(int));
    int *voisins = (int*)malloc(nbVoisinMax*nbCellule()*sizeof(int));
    int s = 0;

    for(int i = 0; i < nbCellule(); i++){
        int nbVoisin = getCellule(i)->nbVoisin();
        pos[i*2] = s;
        pos[i*2+1] = s + nbVoisin -1;

        for(int j = 0; j < nbVoisin; j++){
            voisins[s+j] = getCellule(getCellule(i)->getVoisin(j));
        }

        s = s + nbVoisin;
    }

    hipMalloc(&d_posVoisins, nbCellule()*2*sizeof(int));
    hipMalloc(&d_voisins,nbVoisinMax*nbCellule()*sizeof(int));

    hipMemcpy(d_posVoisins, pos, nbCellule()*2*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_voisins, voisins, nbVoisinMax*nbCellule()*sizeof(int), hipMemcpyHostToDevice);

    free(pos);
    free(voisins);
}

void Graphe::setVoisinsProche(int nbVoisins){
    t = typeVoisinage::PLUSPROCHE;
    nbVoisinMax = nbVoisins*2;
    calculPPV(nbVoisins);
    setVoisinCUDA();
}

void Graphe::resetVoisinPlusProche(){

    cout << "reset voisinage plus proche" << endl;
    for (int i = 0; i < this->nbCellule(); i++) {
        this->getCellule(i)->resetVoisin();
    }

    this->setVoisinsProche(nbVoisinMax);
}

void Graphe::calculPPV(int nbVoisins){

    int* plusProche = (int*)malloc(sizeof(int)*nbVoisins);
    auto* dist = (double*)(malloc(sizeof(double) * nbVoisins));

    for (int i = 0; i < nbCellule();i++){;
        getCellule(i)->setNbVoisinMax(typeVoisinage::PLUSPROCHE);
        for (int j = 0; j < nbVoisins; j++){
            dist[j] = numeric_limits<int>::max();;
        }

       for (int j = 0; j < nbCellule();j++){ 
            if (i != j){
                double distIJ = sqrt(pow((getCellulePosX(i) - getCellulePosX(j)),2) +
                                   pow((getCellulePosY(i) - getCellulePosY(j)),2));
                int max = numeric_limits<int>::min();
                int indiceMax = -1;

                for (int k =0; k < nbVoisins; k++){
                    if (dist[k] > max){
                        max = dist[k];
                        indiceMax = k;
                    }
                }
                if( max > distIJ){
                    plusProche[indiceMax] = j;
                    dist[indiceMax] = distIJ;
                }
            }
       }
       for (int j = 0; j < nbVoisins; j++){
            getCellule(i)->addVoisin(getCellule(plusProche[j]));
            //getCellule(plusProche[j])->addVoisin(getCellule(i));
       }
    }

    free(plusProche);
    free(dist);
}

/*
fonction des différentes mise à jours du graphe
*/

/*void Graphe::contamination() {
    for (int i = 0; i < this->nbCellule();i++)
            this->getCellule(i)->contamination();

    for (int i = 0; i < this->nbCellule();i++)
        this->getCellule(i)->nextT();
}
*/
void Graphe::MAJ() {
    if(c == typeCellule::QUOROMD){
        auto* cell = dynamic_cast<CelluleQuoromD*>(getCellule(0));
        double lambda = cell->getLambda();
        *alerte = 0;

        hipMemcpy(d_nbAlerte, alerte, sizeof(int), hipMemcpyHostToDevice);

        QuorumD<<<1,size>>>(rand(),d_cels, d_celsSync, d_posVoisins, d_voisins,d_nbAlerte,lambda);
        hipDeviceSynchronize();

        hipMemcpy(alerte, d_nbAlerte, sizeof(int), hipMemcpyDeviceToHost);

        cout << "nb Alerte : " << *alerte << endl;

        int *C = (int*)malloc(size*sizeof(int));
        hipMemcpy(C, d_celsSync, size*sizeof(int), hipMemcpyDeviceToHost);

        for(int i = 0; i < nbCellule(); i++){
            if(C[i] == DEFAILANTE) getCellule(i)->setEtat(DEFAILANTE);
            else if(C[i] == ALERTE) getCellule(i)->setEtat(ALERTE);
            else getCellule(i)->setEtat(NORMAL);
        }

        hipMemcpy(d_cels, C, size*sizeof(int), hipMemcpyHostToDevice);

        threshold = (double)*alerte / ((double)(size - nbDef()));
    }
    else{
        #pragma omp parallel for
        for (int i = 0; i < this->nbCellule(); i++) {
            auto* cell = dynamic_cast<CelluleQuoromD*>(getCellule(i));
            cell->transition();
        }

        double nbDef = 0;
        double nbAlerte = 0;

        #pragma omp parallel for shared(nbDef,nbAlerte)
        for (int i = 0; i < this->nbCellule(); i++) {
                this->getCellule(i)->nextT();
                if (this->getCellule(i)->getEtat() == ALERTE) nbAlerte++;
                else if (this->getCellule(i)->getEtat() == DEFAILANTE) nbDef++;
        }

        threshold = nbAlerte / ((double)(size - nbDef));
    }
}

/* 
getter setter
*/

//Cellule *Graphe::getCellule(int i) { return get<0>(this->posCellule.at(i)); }
Cellule* Graphe::getCellule(int i) { return cels[i]; }

int Graphe::getCellule(Cellule *c){
    for (int i = 0; i< nbCellule(); i++){
        if(cels[i] == c) return i;
    }
    return -1;
}

int Graphe::getCellulePosX(int i) { return get<0>(posCellule.at(i)); }
int Graphe::getCellulePosY(int i) { return get<1>(posCellule.at(i)); }

void Graphe::copyCels(){
    int* copy = (int*)malloc(nbCellule()*sizeof(int));
    for (int i = 0; i < nbCellule(); i++) copy[i] = getCellule(i)->getEtat();
    hipMemcpy(d_cels, copy, nbCellule()*sizeof(int), hipMemcpyHostToDevice);
    free(copy);
}

void Graphe::setCelluleDef(double proba) {
    this->threshold = 0;
    for (int i = 0; i < this->nbCellule(); i++) {
        double r = rand() / double(RAND_MAX);
        if (r < proba) {
            this->getCellule(i)->setEtat(DEFAILANTE);
            def++;
        } else
            this->getCellule(i)->setEtat(NORMAL);
    }
    copyCels();
}


bool Graphe::seuil(double s) {
    if (threshold >= s)return true;
    else return false;
}

int Graphe::nbAlerte() {
    int s= 0;
    for (int i = 0; i < this->nbCellule(); i++)
        if (getCellule(i)->getEtat() == ALERTE) s++;
    return s;
}

int Graphe::nbDef() {
    return def;
}


int Graphe::nbCellule() {
    return this->size;
}

void Graphe::setLambda(double lambda) {
    if (this->c == typeCellule::QUOROMD){
        for (int i = 0; i < this->nbCellule(); i++){
            auto* cell = dynamic_cast<CelluleQuoromD*>(getCellule(i));
            cell->setLambda(lambda);
        }
    }
}
/*
void Graphe::setPreCalcul(double *preCalcul) {
    if (this->c == typeCellule::QUOROMD){
        for (int i = 0; i < this->nbCellule(); i++){
            auto* cell = dynamic_cast<CelluleQuoromD*>(getCellule(i));
            cell->setPreCalcul(preCalcul);
        }
    }
}

int Graphe::getHeight(){
    return this->height;
}


int Graphe::getWidth(){
    return this->width;
}

pair<int, int> Graphe::getPosCellule(int i) {
    return pair<int,int>(-1,-1);
}


int Graphe::getPosCelluleX(Cellule *c) {
    for (int i = 0; i < this->nbCellule(); i++){
       if (this->getCellule(i) == c){
            return getCellulePosX(i);
       }
    }
    return -1;
}

int Graphe::getPosCelluleY(Cellule *c) {
    for (int i = 0; i < this->nbCellule(); i++){
       if (this->getCellule(i) == c){
            return getCellulePosY(i);
       }
    }
    return -1;
}
*/

typeVoisinage Graphe::getVoisinage() {
    return this->t;
}