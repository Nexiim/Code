#include "hip/hip_runtime.h"
#include "MAJQuoromD.cuh"

__global__  void QuorumD(int seed, int *cels, int *celsSync, int *posVoisins, int *voisins,int *nbAlerte,double lambda){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    hiprandState localstate;
    hiprand_init(seed,i,0,&localstate);

    if (cels[i] == DEFAILANTE){
        celsSync[i] = DEFAILANTE;
        return;
    }
    
    int nbVoisinN = 0;
    int nbVoisinAD = 0;
    int nbVoisinT = 0;

    for (int a = posVoisins[i*2]; a <= posVoisins[i*2+1]; a++){
        if (voisins[a] == -1) break;
        if(cels[voisins[a]] == NORMAL) nbVoisinN++;
        else nbVoisinAD++;
        nbVoisinT++;
    }
    
    if (nbVoisinN == nbVoisinT){
        celsSync[i] = NORMAL;
        return;
    }

    if (nbVoisinAD == nbVoisinT) {
        celsSync[i] = ALERTE;
        atomicAdd(nbAlerte,1);
        return;
    }
    
    double wn = exp(lambda * ((double)nbVoisinN / (double)nbVoisinT));
    double wad = exp(lambda * ((double)nbVoisinAD / (double)nbVoisinT));
    double probaN = wn / (wn + wad);

    float r = hiprand_uniform(&localstate);

    if(r < probaN )celsSync[i] = NORMAL;
    else {
        atomicAdd(nbAlerte,1);
        celsSync[i] = ALERTE;
    }
    return;
}


